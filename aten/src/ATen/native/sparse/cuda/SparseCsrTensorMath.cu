#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/ExpandUtils.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/NativeFunctions.h>
#include <ATen/SparseCsrTensorImpl.h>
#include <ATen/SparseCsrTensorUtils.h>
#include <ATen/SparseTensorUtils.h>
#include <ATen/WrapDimUtilsMulti.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/Resize.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <type_traits>

#include <THC/THCThrustAllocator.cuh>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAUtils.h>
#include <c10/cuda/CUDACachingAllocator.h>

#include <ATen/native/sparse/cuda/SparseBlasImpl.h>
#include <ATen/native/sparse/cuda/SparseCUDABlas.h>
#include <ATen/native/sparse/cuda/SparseCUDATensorMath.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>

namespace at {
namespace native {

namespace {

template <typename input_t, typename output_t>
__global__ void convert_indices_from_coo_to_csr_cuda_kernel(output_t* data_out, const input_t* data_in, const int64_t size, const int64_t numel) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid == 0) {
    for (int64_t i = 0; i <= data_in[0]; i++)
      data_out[i] = static_cast<output_t>(0);
  } else if (tid < numel) {
    for (int64_t i = data_in[tid - 1]; i < data_in[tid]; i++)
      data_out[i + 1] = static_cast<output_t>(tid);
  } else if (tid == numel) {
    for (int64_t i = data_in[numel - 1] + 1; i < size + 1; i++)
      data_out[i] = static_cast<output_t>(numel);
  }
}

template <typename input_t, typename output_t>
void convert_indices_from_coo_to_csr_cuda(const Tensor& result, const Tensor& input, const int64_t size) {
  int64_t numel = input.numel();
  const input_t* data_in = input.data_ptr<input_t>();
  output_t* data_out = result.data_ptr<output_t>();

  if (numel == 0) {
    result.zero_();
    return;
  }

  // Run (numel + 1) threads...
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (numel + THREADS) / THREADS;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  convert_indices_from_coo_to_csr_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(data_out, data_in, size, numel);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

} // namespace

using namespace at::sparse_csr;
// certain utiliy functions are usable from sparse COO.
using namespace at::sparse;

Tensor& addmm_out_sparse_csr_dense_cuda(
    const Tensor& self,
    const SparseCsrTensor& mat1,
    const Tensor& mat2,
    const Scalar& beta,
    const Scalar& alpha,
    Tensor& result) {

  TORCH_INTERNAL_ASSERT(mat1.is_sparse_csr());

  // All the checks are from addmm_out_cuda_impl at ATen/native/cuda/Blas.cpp
  // TODO: remove code duplication and unify code
  // There were undefined symbol problems,
  // when using same function for CUDA and SparseCsrCUDA dispatch keys
  TORCH_CHECK(mat1.dim() == 2 && mat2.dim() == 2, "tensors must be 2-D");

  TensorArg args[]{{result, "out", 0}, {self, "self", 1}, {mat1, "mat1", 2}, {mat2, "mat2", 3}};
  checkAllSameGPU(__func__, args);

  IntArrayRef mat1_sizes = mat1.sizes();
  IntArrayRef mat2_sizes = mat2.sizes();
  IntArrayRef self__sizes;
  c10::MaybeOwned<Tensor> self_;
  if (&result != &self) {
    self_ = expand_size(self, {mat1_sizes[0], mat2_sizes[1]}, "addmm");
    self__sizes = self_->sizes();
  } else {
    self_ = c10::MaybeOwned<Tensor>::borrowed(self);
    self__sizes = self_->sizes();
    TORCH_CHECK(result.dim() == 2, "tensors must be 2-D");
    TORCH_CHECK(self__sizes[0] == mat1_sizes[0], "self_ dim 0 must match mat1 dim 0");
    TORCH_CHECK(self__sizes[1] == mat2_sizes[1], "self_ dim 1 must match mat2 dim 1");
  }

  if (&result != &self) {
    at::native::resize_output(result, self__sizes);
    if (beta.toComplexDouble() != 0.0) {
      at::native::copy_(result, *self_);
    }
  }

  IntArrayRef result_sizes = result.sizes();
  if ((result_sizes[0] == 0) || (result_sizes[1] == 0)) {
    return result;
  }

  if (mat1._nnz() == 0) {
    // By definition, when beta==0, values in self should be ignored. nans and infs
    // should not propagate
    if (beta.toComplexDouble() == 0.) {
      return result.zero_();
    }
    return at::mul_out(
        result,
        self,
        at::native::scalar_tensor(
            beta,
            self.scalar_type(),
            c10::nullopt /* layout */,
            at::kCPU,
            c10::nullopt /* pin_memory */));
  }

  sparse::impl::cuda::addmm_out_sparse_csr(mat1, mat2, beta, alpha, result);
  return result;
}

Tensor& add_out_dense_sparse_csr_cuda(
    Tensor& output,
    const Tensor& dense,
    const SparseCsrTensor& src,
    const Scalar& alpha) {
  TORCH_INTERNAL_ASSERT(dense.layout() == kStrided);
  TORCH_INTERNAL_ASSERT(src.is_sparse_csr());
  TORCH_INTERNAL_ASSERT(dense.is_cuda());

  TORCH_CHECK(
      output.is_contiguous(),
      "out argument must be contiguous, but got: ",
      output.suggest_memory_format());
  TORCH_CHECK(
      output.is_cuda(),
      "add: expected 'out' to be CUDA tensor, but got tensor on device: ",
      output.device());

  TORCH_CHECK(
      src.is_cuda(),
      "add: expected 'other' to be a CUDA tensor, but got tensor on device: ",
      src.device());

  TORCH_CHECK(
      dense.sizes().equals(src.sizes()),
      "add: expected 'self' and 'other' to have same size, but self has size ",
      dense.sizes(),
      " while other has size ",
      src.sizes(),
      " (FYI: dense-sparse addition does not currently support broadcasting)");

  auto commonDtype = promoteTypes(dense.scalar_type(), src.scalar_type());
  TORCH_CHECK(
      canCast(commonDtype, output.scalar_type()),
      "Can't convert result type ",
      commonDtype,
      " to output ",
      output.scalar_type(),
      " in add operation");

  Tensor src_values = src.values();
  Tensor src_crow_indices = src.crow_indices();
  Tensor src_col_indices = src.col_indices();

  resize_output(output, dense.sizes());

  Tensor resultBuffer = output;
  Tensor valuesBuffer = src_values.to(commonDtype);
  if (output.scalar_type() != commonDtype) {
    resultBuffer = dense.to(commonDtype);
  } else if (!is_same_tensor(output, dense)) {
    resultBuffer.copy_(dense);
  }
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kHalf, kBool, kBFloat16,
      commonDtype,
      "add_out_op2_sparse_csr",
      [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
        AT_DISPATCH_INDEX_TYPES(
            src_crow_indices.scalar_type(),
            "csr_add_out_crow_indices",
              [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
                scalar_t* values_accessor = valuesBuffer.data_ptr<scalar_t>();
                scalar_t* out_ptr = resultBuffer.data_ptr<scalar_t>();
                scalar_t cast_value = alpha.to<scalar_t>();

                index_t* crow_indices_accessor = src_crow_indices.data_ptr<index_t>();
                index_t* col_indices_accessor = src_col_indices.data_ptr<index_t>();
                int64_t out_storage_offset = resultBuffer.storage_offset();

                auto out_strides = resultBuffer.strides();
                int64_t out_strides0 = out_strides[0];
                int64_t out_strides1 = out_strides[1];

                hipStream_t stream = at::cuda::getCurrentCUDAStream();
                auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
                auto policy = thrust::cuda::par(allocator).on(stream);

               // Note that this could be wildly imbalanced if the sparsity pattern varies a lot between rows.
               thrust::for_each(
                    policy,
                    thrust::make_counting_iterator(int64_t(0)),
                    thrust::make_counting_iterator(int64_t(src_crow_indices.size(0) - 1)),
                    [values_accessor,
                    crow_indices_accessor,
                    col_indices_accessor,
                    out_ptr,
                    out_storage_offset,
                    out_strides0,
                    cast_value,
                    out_strides1
                    ]__device__(int64_t irow) {
                        index_t start_index = crow_indices_accessor[irow];
                        index_t end_index = crow_indices_accessor[irow + 1];

                        for (index_t i = start_index; i < end_index; ++i) {
                            auto icol = col_indices_accessor[i];
                            auto index = out_storage_offset + irow * out_strides0 + icol * out_strides1;
                            out_ptr[index] += cast_value * values_accessor[i];
                        }
                    });
              });
      });
  if (output.scalar_type() != commonDtype) {
    output.copy_(resultBuffer);
  }
  return output;
}

Tensor& add_out_sparse_csr_cuda(
    const Tensor& self,
    const SparseCsrTensor& other,
    const Scalar& alpha,
    SparseCsrTensor& out) {
  if (self.layout() == kStrided) {
    return add_out_dense_sparse_csr_cuda(out, self, other, alpha);
  } else {
    TORCH_CHECK(
        false,
        "NotImplementedError: Addition of sparse CSR tensors is not yet implemented.")
  }
  return out;
}

TORCH_IMPL_FUNC(_convert_indices_from_coo_to_csr_structured_cuda) (
  const Tensor& input, const int64_t size, const bool out_int32, const Tensor& result
) {
  if (out_int32) {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int>(result, input, size);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int64_t>(result, input, size);
    });
  }
}

} // namespace native
} // namespace at
